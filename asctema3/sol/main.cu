#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include "helper.h"
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

vector<Input> take_in(const char* fileIn)
{
    string geon1;
    float lat1;
    float lon1;
    int pop1;
    vector<Input> in;
    ifstream ifs(fileIn);

    while(ifs >> geon1 >> lat1 >> lon1 >> pop1) {

        Input aux;
        aux.geon = geon1;
        aux.lat = lat1;
        aux.lon = lon1;
        aux.pop = pop1;

        in.push_back(aux);
    }

    ifs.close();

    return in;
}

__device__ float geoDistances(float lat1, float lon1, float lat2, float lon2)
{
	float phi1 = (90.f - lat1) * DEGREE_TO_RADIANS;
    	float phi2 = (90.f - lat2) * DEGREE_TO_RADIANS;
    	float theta1 = lon1 * DEGREE_TO_RADIANS;
    	float theta2 = lon2 * DEGREE_TO_RADIANS;
	float cs = sin(phi1) * sin(phi2) * cos(theta1 - theta2) + cos(phi1) * cos(phi2);
    
	if (cs > 1) {
       		cs = 1;
    	} else if (cs < -1) {
       		cs = -1;
    	}

    	return 6371.f * acos(cs);
}
__global__ void calc_pop(const float *lat, const float *lon, const int *pop, int *acc_pop, const size_t km_range, const size_t n)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < n && j < n) {
		if(i < j) {

			float distance = geoDistances(lat[i], lon[i], lat[j], lon[j]);
			if(distance <= km_range)
			{
				atomicAdd(&acc_pop[i], pop[j]);
				atomicAdd(&acc_pop[j], pop[i]);
			}
                }
                else if(i == j)
                {
                        atomicAdd(&acc_pop[i], pop[j]);
                }
        }

}

int main(int argc, char** argv) {
    DIE( argc == 1,
         "./accpop <kmrange1> <file1in> <file1out> ...");
    DIE( (argc - 1) % 3 != 0,
         "./accpop <kmrange1> <file1in> <file1out> ...");

    for(int argcID = 1; argcID < argc; argcID += 3) {
        float kmRange = atof(argv[argcID]);
        vector<Input> in;

	in = take_in(argv[argcID + 1]);	       
	ofstream ofs(argv[argcID + 2]);
	float *lat = 0;
	float *lon = 0;
	int *pop = 0;
	int *acc_pop = 0;
	int num_elem = in.size();

	hipMallocManaged(&lat, num_elem * sizeof(float));
	hipMallocManaged(&lon, num_elem * sizeof(float));
	hipMallocManaged(&pop, num_elem * sizeof(int));
	hipMallocManaged(&acc_pop, num_elem * sizeof(int));
	
	if (lat == 0 || lon == 0 || pop == 0 || acc_pop == 0) {
        	cout << "[HOST] Couldn't allocate memory\n";
        	return 1;
    	}
	
	for(int i = 0; i < num_elem; i++)
       	{
		lat[i] = in[i].lat;
		lon[i] = in[i].lon;
		pop[i] = in[i].pop;
		acc_pop[i] = 0;
	}
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 block(32,32);
	dim3 grid((num_elem + 31)/32,  (num_elem + 31)/32);

	hipEventRecord(start);
	calc_pop<<<grid, block>>>(lat, lon, pop, acc_pop, kmRange, num_elem);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	/*
       	float ms = 0;
	hipEventElapsedTime(&ms, start, stop);
	float seconds = ms / pow((float) 10, 3);

	cout<< "time = " << seconds  << endl;
	*/
	for(int i = 0; i < num_elem; i++)
		ofs << acc_pop[i] << endl;

	hipFree(lat);
	hipFree(lon);
	hipFree(pop);
	hipFree(acc_pop);
    }
}
